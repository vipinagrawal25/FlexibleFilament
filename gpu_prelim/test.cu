#include "hip/hip_runtime.h"
#include <stdio.h>
#include<stdlib.h>
#include <math.h>
//#include "evolve.h"
//#include "chain.h"
#include "hip/hip_runtime.h"
#define   nn 10
struct DM{
  double x;
};
int size_DM = sizeof(double);
/* ----------------------------------------------------- */
__device__ void increment(double *psi, DM *a, int kth ){
  psi[kth] = psi[kth] + (*a).x ;
}
__global__ void kernel( double *psi, DM *a){
   int tid = threadIdx.x + blockIdx.x * blockDim.x ;
   if ( tid < nn ){
     increment( psi, a, tid ) ;
   }
}
/* ----------------------------------------------------- */
struct EV  TT;
void test_struct( void );
int main( void ){
  hipDeviceProp_t *prop;
  double PSI[nn] ;
  double *dev_psi;
  // double X;
  //double *dev_x;
  DM A;
  DM *dev_a ;
  printf( "psi[i]= \t");
  for(int i=0; i<nn; i++) {
    PSI[i] = (double) i;
    printf( "%f\t", PSI[i] );
  }
  printf( "\n" );
  hipMalloc( (void**)&dev_psi, nn*sizeof( double ) );
  hipMemcpy( dev_psi, &PSI, nn*sizeof(double), hipMemcpyHostToDevice );
  A.x = 10.;
  hipMalloc( (void**)&dev_a, size_DM );
  hipMemcpy( dev_a, &A, size_DM, hipMemcpyHostToDevice );
  kernel<<<nn,1>>> (dev_psi, dev_a) ;
  hipMemcpy( &PSI, dev_psi, nn*sizeof(double), hipMemcpyDeviceToHost );
  printf( "psi[i]= \t");
  for(int i=0; i<nn; i++) {
    printf( "%f\t", PSI[i] );
  }
  printf( "\n" );
}
/*---------------------------------*/

