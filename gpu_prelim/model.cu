#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "odeN.h"
#include "model.h"
__host__ void write_param( void );
struct MPARAM host_param ;
struct MPARAM *dev_param;
double *DIAG ;
double *dev_diag ;
/* ========================================= */
void set_param( void ){
  host_param.height = 1.;
  double height = host_param.height ;
  host_param.aa = height/(double)(NN-1);
  // distance between two nodes.
  host_param.Dbyell = 0.005 ;
  host_param.dd = height*host_param.Dbyell ;
  /* r/l ratio for the rod has been kept constant. It should be noted that 
     the particles would also have same diameter. */
  host_param.viscosity = 10;	      // Equivalent to kinematic viscosity of glycerin
  host_param.Z0=0. ;		      // If we want the bottom point of the rod to be fixed.
  host_param.FFZ0 = 0. ;	      // Different force for different configuration.
  // Sigma is a dimensionless number, which is described as frequency parameter.
  host_param.sigma=1.5;					
  host_param.ShearRate = 1.;
  host_param.omega = ShearRate*sigma;
  //
  host_param.factorAA = 0.15 ; 
  host_param.AA = host_param.factorAA*pow(10,-4) ; // AA is the bending rigidity.
  host_param.KK = 64.;
  double asqr = host_param.aa*host_param.aa ;
  host_param.HH = host_param.KK*host_param.AA/( asqr );
 // Follow: bit.ly/2r23lmA unit -> Pa.m^4/m^2 -> Pa.m^2
// double TMAX = ShearRate*10;
// double tdiag = TMAX/2000;
  host_param.qdiag = 2 ;
  qdiag = host_param.qdiag ;
  hipMemcpy( dev_param, &host_param,
                     size_MPARAM, hipMemcpyHostToDevice ) ;
  write_param( );
  // allocate host data for diagnostic
  int size_diag = NN*qdiag ; 
  *DIAG = (double *) malloc( sizeof(double)*size_diag ) ;
  hipMalloc( (void**)&dev_diag, size_diag*sizeof( double ) );
}
/* ===================================== */
void write_param( void ){
  printf( "# =========== Model Parameters ==========\n" );
  printf( " #Model : Elastic String \n " ) ;
  printf( "#dimension of ODE:\n pp =  %d \n", pp ) ;
  printf( "#Number of copies:\n  NN = %d\n", NN ) ;
  printf( " height = %f \n" , host_param.height) ;	
  // complete writing out the rest. 
  /*   double aa; 	// distance between two nodes.
  double Dbyell // diameter/length of the filament.
  double dd ;	/* r/l ratio for the rod has been kept constant. 
                   It should be noted that the particles would also have same diameter. */
  /*double viscosity ;				
  double  Z0;	  // If we want the bottom point of the rod to be fixed.
  double FFZ0 ; // Force Value on the ends
// Sigma is a dimensionless number, which is described as frequency parameter.
  double sigma ;					
  double ShearRate ;
  double omega ;
  double  factorAA ; 
  double AA ;
  double HH ;		// Follow: bit.ly/2r23lmA unit -> Pa.m^4/m^2 -> Pa.m^2
  double KK; */
  printf( " #============================\n" );
}
/* ===================================== */
__device__ void eval_rhs( double dpsi[], double psi[], int kelement, double tau,
                          struct MPARAM *param, double *diag ){

  vec3 R, dR, EForce, FF0;  // R is the position of the beads.
  /* we are calculating two diagnostic quantities at the moment
ds : d( material coordinate) . 
kappasqr : square of local curvature. 
This number is stored in param.qdiag */
  double qdiag = param.qdiag ;
  double ds, kappasqr ; 
  double onebythree = 1./3.;
  double Curvlength = 0; // length of the filament
  R.x= psi[pp*kelement];
  R.y= psi[pp*kelement + 1];
  R.z= psi[pp*kelement + 2];
  Rp1.x= psi[(pp+1)*kelement];
  Rp1.y= psi[(pp+1)*kelement + 1];
  Rp1.z= psi[(pp+1)*kelement + 2];
  ds = norm( Rp1-R); // this is not correct yet
  dHdR( kelement, R, &EForce, &kappasqr );

  /* write diagnostic to corresponding array */
  diag[kelement*qdiag +1] = ds ;
  diag[kelement*qdiag +1]  = kappasqr;
  drag(R, dR, EForce);

  switch(conf_number){
    case 0:
      FF0.x = 0;
      FF0.y = 0;
      FF0.z = -FFZ0*sin(omega*time);
      EForce[Np-1] = EForce[Np-1]-FF0;
      break;

    case 1:
      for (int ip = 0; ip < Np; ++ip)
      {
        if (sin(omega*time) >= 0){
          dR[ip].y = dR[ip].y + ShearRate*(height - R[ip].z)*ceil(sin(omega*time));    
        }
        else{
          dR[ip].y = dR[ip].y + ShearRate*(height - R[ip].z)*floor(sin(omega*time));
        }
      }
      break;

      case 2:
      for (int ip = 0; ip < Np; ++ip)
      {
        dR[ip].y = dR[ip].y + ShearRate*(R[ip].z);          
      }
      break; 
  }
  
  // External force applied on the end point.
  // cout << FF0.z <<endl;
  // dR[Np-1] = dR[Np-1]-FF0*; 
  //dR[Np-1].y = 0;                     // Constraint that last point should always remain on z axis. 
  
  dpsi[pp*kelement] = dR.x  ;
  dpsi[pp*kelement + 1] = dR.y ;
  dpsi[pp*kelement + 2] = dR.z ;
/**************************--------------------------------------------------------------*/
void drag(vec3 X[], vec3 dX[], vec3 EForce[]){
  double onebythree = 1./3.;
  double mu0 = onebythree/(M_PI*viscosity*dd);
  if (UseRP == 'Y')
  {
    // mu_ij represents the one element of mobility matrix (Size: NXN). 
    // Every element of the matrix itself is a 2nd rank tensor and the dimension of that should 3x3.

    Tens2 mu_ij, mu_ii;
    double d_rij;
    vec3 rij;

    mu_ii = dab*mu0;    // dab is the unit 2 dimensional tensor. It is defined in module/2Tens file.
    // PTens2(mu_ii);
    // rij = R[j]-R[i] and d_rij is just the norm of this value.

    for (int ip = 0; ip < Np; ++ip)
    {
        // The mu_ij in the next line represents the mobility tensor when j is equal to i and in 
        // response to that the "for loop" is started from ip+1 .

        dX[ip] = dX[ip] + dot(mu_ii, EForce[ip]);
        
        for (int jp = ip+1; jp < Np; ++jp)
        {
            GetRij(X, ip, jp, &d_rij, &rij);
            double c1 = 1/(8*M_PI*viscosity*d_rij);
            double dsqr1 = 1./(d_rij*d_rij);
            mu_ij = c1*(dab + (rij*rij)*dsqr1 + dd*dd/(2*d_rij*d_rij)*(dab*onebythree - (rij*rij)*dsqr1));
            dX[ip] = dX[ip] + dot(mu_ij, EForce[ip]);
            dX[jp] = dX[jp] + dot(mu_ij, EForce[ip]);
        }
    }
  }

  else
  {
    for (int ip = 0; ip < Np; ++ip)
    {
        dX[ip] = EForce[ip]*mu0;
    }
  } 
 
}

/**************************/
void getub(double *bk, vec3 *uk, int kp, vec3 X[]){
  vec3 dX = X[kp+1]-X[kp];
  double bb = norm(dX);
  *bk = bb;
  *uk =dX/bb;
}
/**************************/
void dHdR(int kp, vec3 X[], vec3* add_FF, double* add_kappasqr ){
     // This function calculates the force at every node which is a function of X, time.
  vec3 ukm2(0.,0.,0.), ukm1(0.,0.,0.), uk(0.,0.,0.), ukp1(0.,0.,0.), Xzero(0.,0.,0.), dX(0.,0.,0.);
  double bkm2, bkm1, bk, bkp1;
  vec3 FF = *add_FF;             
 
  
 

  switch(kp){
    case 0:
      getub(&bk, &uk, kp, X);
      getub(&bkp1, &ukp1, kp+1, X);
      if (conf_number==0 ){
          dX = X[kp-1+1]-Xzero;
          bkm1 = norm(dX);
          // cout << bkm1 << endl;
          ukm1=dX/bkm1;
          FF = (     (uk)/bkm1 - (ukm1+ukp1)/bk
               + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
               - (ukm1/bkm1)*( dot(ukm1,uk) )
               );
          FF = FF*AA/aa;
          // Add an extra term for inextensibility constraint
          FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa; 
          // cout << FF.z << endl;
          *add_FF = FF;
          // *add_SS = (kp+1)*bkm1;
      }
      else{
          FF = ( (uk/bk)*( dot(uk,ukp1) )  - (ukp1)/bk );
          FF = FF*AA/aa;
          // Add an extra term for inextensibility constraint
          FF = FF + ( uk*(bk-aa))*HH/aa; 
          // cout << FF.z << endl;
          *add_FF = FF;
          // cout << "Kya ye yaha aa raha hai?" << endl;
          // *add_SS = 0;
          // cout << bk << '\t' << aa << endl;  

          break;
      }
      
      *add_kappasqr=0.;
      break;     

    case 1:

      getub(&bkm1, &ukm1, kp-1, X);
      getub(&bk, &uk, kp, X);
      getub(&bkp1, &ukp1, kp+1, X);
      
      if (conf_number==0)
      {
          dX = X[kp-2+1]-Xzero;
          bkm2 = norm(dX);
          ukm2 = dX/bkm2;
          FF = (  (uk+ukm2)/bkm1 - (ukm1+ukp1)/bk
              + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
              - (ukm1/bkm1)*( dot(ukm1,ukm2) + dot(ukm1,uk) )
              );
          FF = FF*(AA/aa);
          // cout << FF.z << endl;
          FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa) )*HH/aa;   // Inextensibility constraint
          *add_FF = FF;
      }
      else
      {
          FF = (     (uk)/bkm1 - (ukm1+ukp1)/bk
              + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
              - (ukm1/bkm1)*( dot(ukm1,uk) )
              );
          FF = FF*(AA/aa);
          // cout << FF.z << endl;
          FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa;   // Inextensibility constraint
          // cout << "Good for you " << endl;
          *add_FF = FF;

      }

      *add_kappasqr=0.;
      // *add_SS = (kp+1)*bkm1;      
      break;

    case Np-2:
      getub(&bkm2, &ukm2, kp-2, X);
      getub(&bkm1, &ukm1, kp-1, X);
      getub(&bk, &uk, kp, X);

      FF = (     (uk+ukm2)/bkm1 - (ukm1)/bk
          + (uk/bk)*( dot(uk,ukm1))
          - (ukm1/bkm1)*( dot(ukm1,ukm2) + dot(ukm1,uk) )
          );    
      FF = FF*(AA/aa);
      // cout << bk << endl;
      // cout << FF.z << endl;
      FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa;    // Inextensibility constraint 
      // cout << FF.z << endl;
      *add_kappasqr=0.;
      *add_FF = FF;  
      // *add_SS = (kp+1)*bkm1;
      break;

    case Np-1:
      getub(&bkm2, &ukm2, kp-2, X);
      getub(&bkm1, &ukm1, kp-1, X);
  
      FF = (     (ukm2)/bkm1
        - (ukm1/bkm1)*( dot(ukm1,ukm2) )
        );
      FF = FF*(AA/aa);
      // cout << bkm1 << endl;
      // cout << FF.y << endl;
      FF = FF - (ukm1*(bkm1-aa))*HH/aa;
      // cout << FF.y << endl;
      *add_kappasqr=0.;
      *add_FF = FF;
      // *add_SS = (kp+1)*bkm1;
      break;

    default:
      getub(&bkm2, &ukm2, kp-2, X);
      getub(&bkm1, &ukm1, kp-1, X);
      getub(&bk, &uk, kp, X);
      getub(&bkp1, &ukp1, kp+1, X);
      FF = (     (uk+ukm2)/bkm1 - (ukm1+ukp1)/bk
          + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
          - (ukm1/bkm1)*( dot(ukm1,ukm2) + dot(ukm1,uk) )
          );
      FF = FF*(AA/aa);
      // cout << bkm1 <<endl;
      // cout << FF.y << endl;
      FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa;    // Inextensibility constraint 
      // cout << FF.x << endl;
      // cout << FF.y << endl;     

      if (flag_kappa==false)
      {
        *add_kappasqr=2.*(1.- dot(uk,ukm1))/(aa*aa);
        // cout << kappasqr << endl;
        // cout << "This is also high level shit" << endl;
      }
      *add_FF = FF;
      // *add_SS = (kp+1)*bkm1;
      break;
  }  

  //   if (flag_kappa)
  //   {
  //     *add_kappasqr=2.*(1.-dot(uk,ukm1));
  //     // cout << kappasqr << endl;
  //   }
  //   *add_FF = FF;
  // }
  // *add_SS = (kp+1)*bkm1;
 }
/**************************/

















  double xx, vv;
  // double OM = (*dev_param).OM ;
  double OM=0.;
  // We solve for 1-d harmonic oscillator.


  xx = psi[kelement];
  vv = psi[kelement+1];
  dpsi[kelement] = vv;
  dpsi[kelement+1] = - OM*OM*xx ;
}
/*-------------------------------------------------------------------*/
__host__ void initial_configuration( double PSI[] ){
  double xx, vv;
  for( int s=0; s<NN; s++){
   // A Harmonic Oscillator 
 /* initially all positions are zero */
    xx = 0.;
 /* and all velocities are unity */
    vv = 1.;
    PSI [pp*s ] = xx;
    PSI[pp*s+1] = vv;
  } 
}
/*-------------------------------------------------------------------*/
