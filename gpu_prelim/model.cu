#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "model.h"
#include "3vec.h"
#include "2Tens.h"
using namespace std;
__device__ void dHdR(int kp, double psi[], vec3* add_FF,
                     double* add_kappasqr,  struct MPARAM *param,
                     struct CRASH * );
__device__ vec3 drag(int ip,  double psi[], vec3 EForce[],
                     struct MPARAM *param);
__device__ vec3 ext_flow( int kelement, vec3 R, double tau,
                          struct MPARAM *param  );
__device__ vec3 ext_force( int kelement, vec3 R, double tau,
                           struct MPARAM *param  );
__device__ void GetRij(double psi[], int i, int j, double *Distance,
                       vec3 *rij);
__device__  void getub(double *bk, vec3 *uk, int kp, double psi[]);
__device__ int square_wave(double t, double Tby2) ;
__device__ vec3 Uflow ( vec3 RR, struct MPARAM *param);
__device__ void device_exception( struct CRASH *bug, char mesg[] );
__device__ vec3 psi2R(double psi[], int k);
__device__ void R2psi(double psi[], int k, vec3 R);
/* ========================================= */
void alloc_chain(  double **PSI, double **dev_psi ){
  double *temp;
  *PSI = (double *) malloc( ndim*sizeof( double ) );
  hipMalloc( (void**)&temp, ndim*sizeof( double ) );
  *dev_psi = temp;
}
/* ------------------------------------------------------------------------------*/
int pre_diag( double **DIAG , double **dev_diag, MPARAM PARAM ){
 // allocate host data for diagnostic
  int qdiag = PARAM.qdiag ; 
  int size_diag = NN * qdiag * sizeof(double) ; 
  *DIAG = (double *) malloc( size_diag ) ;
  for (int iN=0; iN<NN; iN++ ){
    for(int iq=0; iq<qdiag; iq++){
      (*DIAG)[iN*qdiag+iq] = 1. ;
    }
  }
  double *temp;
  hipMalloc( (void**)&temp, size_diag );
  *dev_diag = temp ;
  hipMemcpy( *dev_diag,  *DIAG, size_diag,
              hipMemcpyHostToDevice);
  return size_diag;
}
/*--------------------------------------------------------*/
void set_param( MPARAM *PARAM, MPARAM **dev_param ){
  MPARAM *temp;
  double height = 1.;
  (*PARAM).height = height;
  (*PARAM).aa = height/(double)(NN-1);
  // distance between two nodes.
  (*PARAM).Dbyell = 0.005 ;
  (*PARAM).dd = height*(*PARAM).Dbyell ;
  /* r/l ratio for the rod has been kept constant. It should be noted that 
     the particles would also have same diameter. */
  (*PARAM).viscosity = 10;	      // Equivalent to kinematic viscosity of glycerin
  (*PARAM).Z0=0. ;		      // If we want the bottom point of the rod to be fixed.
  (*PARAM).Famp = 0. ;	      // Different force for different configuration.
  // Sigma is a dimensionless number, which is described as frequency parameter.
  (*PARAM).sigma=1.5;					
  (*PARAM).ShearRate = 1.;
  (*PARAM).omega = (*PARAM).ShearRate*(*PARAM).sigma ;
  //
  (*PARAM).factorAA = 0.15 ;
  // (*PARAM).factorAA = 0. ;
  (*PARAM).AA = (*PARAM).factorAA*pow(10,-4) ; // AA is the bending rigidity.
  (*PARAM).KK = 64.;
  double asqr = (*PARAM).aa*(*PARAM).aa ;
  (*PARAM).HH = (*PARAM).KK*(*PARAM).AA/( asqr );
// Follow: bit.ly/2r23lmA unit -> Pa.m^4/m^2 -> Pa.m^2
// double TMAX = ShearRate*10;
// double tdiag = TMAX/2000;
  (*PARAM).qdiag = 2 ;
  int qdiag = (*PARAM).qdiag ;
  (*PARAM).bcb = 1 ;      // Boundary condition at bottom
  (*PARAM).bct = 1;       // Boundary condition at top
  (*PARAM).global_drag = 1; 
  (*PARAM).iext_force = 0; // Whether to apply the external force or not
  (*PARAM).floc = 0 ;       // External force location 
  (*PARAM).iext_flow = 1;   // External flow: Yes/No
  (*PARAM).iniconf = 1;     // Configuration of the system at t = 0.
  hipMalloc( (void**)&temp, size_MPARAM  );
  *dev_param = temp;
  hipMemcpy( *dev_param, PARAM,
                     size_MPARAM, hipMemcpyHostToDevice ) ;
  write_param( PARAM, "wparam.txt" );
} 
  // allocate space for crashing gracefully.
  /* hipMalloc( (void**)&dev_bug, size_CRASH );
  BUG.lstop = 0;
  strcpy(BUG.message, " No bug yet" );
  hipMemcpy( dev_bug, &BUG, size_CRASH, hipMemcpyHostToDevice); */
/* ------------------------------------------------------------------------------*/
__host__ void write_param( MPARAM *PARAM, char *fname ){
  FILE *pout ;
  pout = fopen ( fname, "w" );
  printf( "# =========== Model Parameters ==========\n" );
  printf( " #Model : Elastic String \n " ) ;
  printf( "#dimension of ODE:\n pp =  %d \n", pp ) ;
  printf( "#Number of copies:\n  NN = %d\n", NN ) ;
  printf( " height = %f \n" , (*PARAM).height) ;
  printf( " #============================\n" );
  fprintf( pout, "# =========== Model Parameters ==========\n" );
  fprintf( pout, " #Model : Elastic String \n " ) ;
  fprintf( pout, "#dimension of ODE:\n pp =  %d \n", pp ) ;
  fprintf( pout, "#Number of copies:\n  NN = %d\n", NN ) ;
  fprintf( pout, " height = %f \n" , (*PARAM).height) ;
  fprintf( pout, " aa= %f \n" , (*PARAM).aa ) ;
  fprintf( pout, " Dbyell= %f \n" , (*PARAM).Dbyell ) ;
  fprintf( pout, " dd= %f \n",  (*PARAM).dd ) ;
  fprintf( pout, " viscosity = %f \n ",  (*PARAM).viscosity) ;
  fprintf( pout, " Z0 = %f \n ", (*PARAM).Z0) ;
  fprintf( pout, " Famp = %f \n ", (*PARAM).Famp ) ;
  fprintf( pout, " sigma = %f \n ", (*PARAM).sigma ) ;
  fprintf( pout, " ShearRate = %f \n ", (*PARAM).ShearRate ) ;
  fprintf( pout, " omega = %f \n ", (*PARAM).omega ) ;
  fprintf( pout, " factorAA = %f \n ", (*PARAM).factorAA ) ;
  fprintf( pout, " AA = %f \n ", (*PARAM).AA ) ;
  fprintf( pout, " KK = %f \n ", (*PARAM).KK ) ;
  fprintf( pout, " HH = %f \n ", (*PARAM).HH ) ;
  fprintf( pout, " qdiag=%d\n", (*PARAM).qdiag );
  fprintf( pout, " bcb=%d\n", (*PARAM).bcb );
  fprintf( pout, " bct=%d\n", (*PARAM).bct );
  fprintf( pout, " global_drag=%d\n", (*PARAM).global_drag );
  fprintf( pout, " iext_force=%d\n", (*PARAM).iext_force );
  fprintf( pout, " floc=%d\n", (*PARAM).floc );
  fprintf( pout, " iext_flow=%d\n", (*PARAM).iext_flow );
  fprintf( pout, " iniconf=%d\n", (*PARAM).iniconf );
  fprintf( pout, " #============================\n" );
  fclose( pout );
}
/* -----------------------------------------------------------------------------------*/
__device__ vec3 psi2R(double psi[], int k){
  vec3 Rt;
  Rt.x = psi[3*k]; 
  Rt.y = psi[3*k+1]; 
  Rt.z = psi[3*k+2];
  return Rt;
}
/* -----------------------------------------------------------------------------------*/
__device__ void R2psi(double psi[], int k, vec3 R){
  psi[3*k] = R.x; 
  psi[3*k+1] = R.y; 
  psi[3*k+2] = R.z;
}
/* -----------------------------------------------------------------------------------*/
__device__ int square_wave( double t, double Tby2) {
  int s = t/Tby2 ;
  int sw = -2*(s % 2 ) + 1;
  return sw;
}
/* -----------------------------------------------------------------------------------*/  
__device__ vec3 Uflow ( vec3 RR, struct MPARAM *param){
  double gdot = (*param).ShearRate ;
  vec3 UU( gdot*RR.y, 0., 0. );
  return UU;
}
/* -----------------------------------------------------------------------------------*/
__device__ vec3 ext_force( int kelement, vec3 R, double tau,
                           struct MPARAM *param  ){
  double omega = (*param).omega;
  double Famp = (*param).Famp;
  vec3 FF0;
  /* iext_force : implements external force on the filament
     periodic forcing at  position floc  */
      FF0.x = 0. ;
      FF0.y = 0. ;
      FF0.z = -Famp*sin(omega*tau) ;
      return FF0;
}

/* -----------------------------------------------------------------------------------*/  
__device__ vec3 ext_flow( int kelement, vec3 R, double tau,
                          struct MPARAM *param  ){
  int iext_flow = (*param).iext_flow;
  double ShearRate = (*param).ShearRate;
  double omega = (*param).omega;
  double height = (*param).height;
  vec3 UU ;
  switch( iext_flow ){
  case 1:
    //time-dependent shear U = ( ShearRate*z, 0, 0 ) * square_wave(omega*time) 
    UU.x = (height - R.z)*ShearRate*(double)square_wave( tau, M_PI/omega ) ;
    UU.y = 0. ;
    UU.z = 0;
    break;
  case 2:
    UU.x = R.z*ShearRate ;
    break;
  }
  return UU;
}
/*--------------------------------------------------------------------------------------*/
__device__ vec3 drag(int ip,  double psi[], vec3 *EForce, struct MPARAM *param){
  Tens2 dab(1.,0.,0.,0.,1.,0.,0.,0.,1.);
  vec3 dR(0., 0., 0.) ;
  double viscosity = (*param).viscosity ;
  double dd = (*param).dd;
  double onebythree = 1./3;
  double mu0 = onebythree/(M_PI*viscosity*dd);
  double c1, dsqr1;
  if ( (*param).global_drag ){
    /* mu_ij represents the one element of mobility matrix (Size: NXN). 
       Every element of the matrix itself is a 2nd rank tensor with dimension 3x3.*/
    Tens2 mu_ij, mu_ii;
    double d_rij;
    vec3 rij;
    //
    mu_ii = dab*mu0;
    /* dab is Kroneker delta in 2d. It is defined in module/2Tens file. */
    // PTens2(mu_ii);
    // rij = R[j]-R[i] and d_rij is just the norm of this value.
    for (int jp = 0; jp < NN; ++jp){
      if (jp == ip){
        dR =  dR + dot(mu_ii, *EForce );
      }else{
        GetRij(psi, ip, jp, &d_rij, &rij);
        c1 = 1./(d_rij*8*M_PI*viscosity);
        dsqr1 = 1./(d_rij*d_rij);
        mu_ij = c1*(dab + (rij*rij)*dsqr1 +
                    dd*dd/(2*d_rij*d_rij)*(dab*onebythree - (rij*rij)*dsqr1));
        dR =  dR + dot(mu_ij, *EForce );
      }
    }
  } else{
    /* if we use local drag */
    dR = (*EForce)*mu0;
  } 
  return dR;
}
/*----------------------------------------------------------------------------------------------*/
__device__ void GetRij(double psi[], int i, int j, double *Distance,
                       vec3 *rij){

  vec3 Rj =  psi2R( psi,  j);
  vec3 Ri =  psi2R( psi,  i);

// __device__ void R2psi(double psi[], int k, vec3 R);

//   vec3 Rj (psi[pp*j], psi[pp*j+1], psi[pp*j+2] );
//   vec3 Ri (psi[pp*i], psi[pp*i+1], psi[pp*i+2] );
  /*This calculate the distance at two index i and j on the elastic string*/
  *rij = Rj - Ri;
  double Dis = norm(Rj-Ri); 
  *Distance = Dis;
}
/*--------------------------------------------------------------*/
__device__  void getub(double *bk, vec3 *uk, int kp, double psi[]){
  vec3 X = psi2R( psi, kp ) ;
  vec3 Xp1 = psi2R ( psi, kp+1) ;
  vec3 dX = Xp1-X;
  double bb = norm(dX);
  *bk = bb;
  *uk =dX/bb;
}
/*-----------------------------------------------------------------------------------------------*/
__device__ vec3 Force_FirstPoint( double psi[],
                                  struct MPARAM *param, struct CRASH *bug ){
  double AA = (*param).AA ;
  double aa = (*param).aa ;
  double HH = (*param).HH;
  int bcb = (*param).bcb ;    
  vec3 ukm2(0.,0.,0.), ukm1(0.,0.,0.), uk(0.,0.,0.), ukp1(0.,0.,0.);
  vec3 Xzero(0.,0.,0.), dX(0.,0.,0.);
  double  bkm1, bk, bkp1;
  vec3 Xbot( psi[0], psi[1], psi[2]  );
  vec3 FF;
  getub(&bk, &uk, 0, psi);
  getub(&bkp1, &ukp1, 1, psi);
  switch( bcb ){
  case 0: 
    dX = Xbot - Xzero;
    bkm1 = norm(dX);
    ukm1=dX/bkm1;
        FF = (     (uk)/bkm1 - (ukm1+ukp1)/bk
               + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
               - (ukm1/bkm1)*( dot(ukm1,uk) )
               );
          FF = FF*AA/aa;
          // Add an extra term for inextensibility constraint
          FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa; 
          break;
    case 1: // free
        FF = ( (uk/bk)*( dot(uk,ukp1) )  - (ukp1)/bk );
        FF = FF*AA/aa;
        // Add an extra term for inextensibility constraint
        FF = FF + ( uk*(bk-aa))*HH/aa; 
        break;
  default: // we must crash now.
    device_exception( bug, "NN=0,  bcb not implemented " );
    break;
  }
  return FF;
}
/*-----------------------------------------------------------------------------------------------*/
__device__ vec3 Force_SecondPoint( double psi[],
                                  struct MPARAM *param, struct CRASH *bug ){
  double AA = (*param).AA ;
  double aa = (*param).aa ;
  double HH = (*param).HH;
  int bcb = (*param).bcb ;
  vec3 ukm2(0.,0.,0.), ukm1(0.,0.,0.), uk(0.,0.,0.), ukp1(0.,0.,0.);
  vec3 Xzero(0.,0.,0.), dX(0.,0.,0.);
  double bkm2, bkm1, bk, bkp1;
  vec3 Xbot( psi[0], psi[1], psi[2]  );
  vec3 FF;
  getub(&bkm1, &ukm1, 0, psi);
  getub(&bk, &uk, 1, psi);
  getub(&bkp1, &ukp1, 2, psi);
  switch( bcb ){
    case 0: //clamped
      dX = Xbot-Xzero;
      bkm2 = norm(dX);
      ukm2 = dX/bkm2;
      FF = (  (uk+ukm2)/bkm1 - (ukm1+ukp1)/bk
              + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
              - (ukm1/bkm1)*( dot(ukm1,ukm2) + dot(ukm1,uk) )
              );
      FF = FF*(AA/aa);
      FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa) )*HH/aa;   // Inextensibility constraint
      break;
    case 1: // free
      FF = (     (uk)/bkm1 - (ukm1+ukp1)/bk
                 + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
                 - (ukm1/bkm1)*( dot(ukm1,uk) )
                 );
      FF = FF*(AA/aa);
      FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa;   // Inextensibility constraint
      break;
  default: // any other boundary conditions.
    device_exception( bug, "NN=1, bcb not implemented ");
    break;
  }
  return FF;
}
/*-----------------------------------------------------------------------------------------------*/
__device__ vec3 Force_NNm2( double psi[],
                                  struct MPARAM *param, struct CRASH *bug ){
  double AA = (*param).AA ;
  double aa = (*param).aa ;
  double HH = (*param).HH;
  int bct = (*param).bct ;
  vec3 ukm2(0.,0.,0.), ukm1(0.,0.,0.), uk(0.,0.,0.), ukp1(0.,0.,0.);
  vec3 Xzero(0.,0.,0.), dX(0.,0.,0.);
  double bkm2, bkm1, bk;             
  vec3 FF; 
  getub(&bkm2, &ukm2, NN-4, psi);
  getub(&bkm1, &ukm1, NN-3, psi);
  getub(&bk, &uk, NN-2, psi);
  FF = (     (uk+ukm2)/bkm1 - (ukm1)/bk
             + (uk/bk)*( dot(uk,ukm1))
             - (ukm1/bkm1)*( dot(ukm1,ukm2) + dot(ukm1,uk) )
             );    
  FF = FF*(AA/aa);
  FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa;    // Inextensibility constraint
  return FF;
}
/*-----------------------------------------------------------------------------------------------*/
__device__ vec3 Force_NNm1( double psi[],
                                  struct MPARAM *param, struct CRASH *bug ){
  double AA = (*param).AA ;
  double aa = (*param).aa ;
  double HH = (*param).HH;
  int bct = (*param).bct ;
  vec3 ukm2(0.,0.,0.), ukm1(0.,0.,0.), uk(0.,0.,0.), ukp1(0.,0.,0.);
  vec3 Xzero(0.,0.,0.), dX(0.,0.,0.);
  double bkm2, bkm1;             
  vec3 FF; 
  switch( bct ){
  case 0: // clamped
    device_exception( bug , "element NN-1, bct=0 not implemented "); 
  break;
 case 1: //free 
   getub(&bkm2, &ukm2, NN-3, psi);
   getub(&bkm1, &ukm1, NN-2, psi);
   FF = (     (ukm2)/bkm1
              - (ukm1/bkm1)*( dot(ukm1,ukm2) )
              );
   FF = FF*(AA/aa);
   FF = FF - (ukm1*(bkm1-aa))*HH/aa;
   break;
 default: // any other bct .
   device_exception( bug, "element NN-1, bct not implemented ");
   break;
  }
  return FF;
}
/*-----------------------------------------------------------------------------------------------*/
__device__ vec3 Force_rest( double* add_kappasqr, int kp, double psi[],
                                  struct MPARAM *param, struct CRASH *bug ){
  double AA = (*param).AA ;
  double aa = (*param).aa ;
  double HH = (*param).HH;
  vec3 ukm2(0.,0.,0.), ukm1(0.,0.,0.), uk(0.,0.,0.), ukp1(0.,0.,0.);
  vec3 Xzero(0.,0.,0.), dX(0.,0.,0.);
  double bkm2, bkm1, bk, bkp1;             
  vec3 FF;
  getub(&bkm2, &ukm2, kp-2, psi);
  getub(&bkm1, &ukm1, kp-1, psi);
  getub(&bk, &uk, kp, psi);
  getub(&bkp1, &ukp1, kp+1, psi);
  FF = (     (uk+ukm2)/bkm1 - (ukm1+ukp1)/bk
             + (uk/bk)*( dot(uk,ukm1) + dot(uk,ukp1) )
             - (ukm1/bkm1)*( dot(ukm1,ukm2) + dot(ukm1,uk) )
             );
  FF = FF*(AA/aa);
  FF = FF - (ukm1*(bkm1-aa) - uk*(bk-aa))*HH/aa;    // Inextensibility constraint
  *add_kappasqr=2.*(1.- dot(uk,ukm1))/(aa*aa);
  return FF;
}
/*-----------------------------------------------------------------------------------------------*/
__device__ void dHdR(int kp, double psi[], vec3* add_FF,
                     double* add_kappasqr,  struct MPARAM *param, struct CRASH *bug ){
     // This function calculates the force at every node which is a function of X, time.
  switch(kp){
  case 0: // 0th element of the chain.
    *add_FF = Force_FirstPoint( psi, param, bug );
    *add_kappasqr=0.;
    break;     
  case 1: //1st element of the chain.
    *add_FF = Force_SecondPoint( psi, param, bug ); 
    *add_kappasqr=0.;
    break;
  case NN-2:
    *add_FF = Force_NNm2( psi, param, bug) ; 
    *add_kappasqr=0.;
    break;
  case NN-1:
    *add_FF = Force_NNm1( psi, param, bug );
    *add_kappasqr=0.;
    break;
  /* for all other points */
  default:
    *add_FF = Force_rest( add_kappasqr, kp, psi, param, bug ) ;
    break;
  }  
}
 /* -----------------------------------------------------------------------------------*/
__device__ void model_rhs( double dpsi[], double psi[], int kelement, double tau,
                           struct MPARAM *param, double diag[], CRASH *bug, int ldiag ){
  int iext_flow = (*param).iext_flow ;
  vec3 R, dR, EForce, FF0, Rp1;  // R is the position of the beads.
  int iext_force = (*param).iext_force ;
  int floc = (*param).floc ;
  /* we are calculating two diagnostic quantities at the moment
ds : d( material coordinate) . 
kappasqr : square of local curvature. 
This number is stored in param.qdiag */
  int qdiag = (*param).qdiag ;
  double ds, kappasqr ; 
  R = psi2R(psi, kelement );
  if ( kelement == (NN-1) ){
    ds = 0.;
  } else {
    Rp1 = psi2R(psi, kelement+1) ;
    ds = norm( Rp1-R );
  }
  dHdR( kelement, psi, &EForce, &kappasqr, param, bug );
  /* write diagnostic to corresponding array */
  if (ldiag ){
  diag[ kelement*qdiag ] = ds ;
  diag[kelement*qdiag +1]  = kappasqr;
  }
  /* add external force to the filament */
  if ( (iext_force) && (kelement == floc) ){
  EForce = EForce -  ext_force( kelement, R, tau, param ) ;}
  /* calculate the viscous (possibly non-local ) drag */
  dR = drag(kelement, psi,  &EForce, param);
  /* contribution from external flow */
  // if ( iext_flow  ){ 
  //   dR = dR+ext_flow( kelement, R, tau, param ) ; 
  // }
  /*------ put the rhs back to the dpsi array ----- */
  R2psi( dpsi, kelement, dR);
}
/*-------------------------------------------------------------------*/
void initial_configuration( double PSI[], MPARAM PARAM ){
  int iniconf = PARAM.iniconf;
  switch(iniconf){
    case -1:
      // Here the thing goes if you want to start the code from middle.
      // In this case, one more parameter is needed though, the last file.
    case 1:
      /* elastic filament is on a straight line perpendicular to the flow
         with no perturbation.*/
      for (int iN=0; iN<NN; iN++){
        PSI[iN*pp] = 0.;
        PSI[iN*pp + 1] = 0.;
        PSI[iN*pp + 2] = (PARAM.aa)*(double) iN ; 
      }     
  }  
}
/*-------------------------------------------------------------------*/
void  free_chain( double **PSI, double **psi  ){
  free( *PSI );
  hipFree( *psi );
}
/*-------------------------------------------------------------------*/
void H2D(double dev_arr[], double ARR[], int Nsize){
  hipMemcpy( dev_arr, ARR, Nsize*sizeof(double), hipMemcpyHostToDevice);
}
/*-------------------------------------------------------------------*/
void D2H(double ARR[], double dev_arr[], int Nsize){
  hipMemcpy( ARR, dev_arr, Nsize*sizeof(double), hipMemcpyDeviceToHost);
}
/*-------------------------------------------------------------------*/
void wPSI ( double PSI[], double tau ){
  FILE *fp = fopen( "data/PSI", "a" );
  fprintf( fp, "%f\t", tau ) ;
  for ( int ichain = 0; ichain< ndim; ichain++ ){ 
    fprintf( fp, "%f\t", PSI[ichain] ) ; 
  }
  fprintf( fp, "\n " );
  fprintf(fp, "#--------------------------------------#\n");
  fclose( fp );
}
/*-------------------------------------------------------------------*/
void wDIAG( double DIAG[], double tau, MPARAM PARAM ){
  FILE *fp = fopen( "data/DIAG", "a" );
  for (int idiag = 0; idiag < PARAM.qdiag ; idiag++){
    fprintf( fp, "%f\t", tau );
    for ( int iN = 0; iN< NN; iN++ ){ 
      fprintf( fp, "%f\t", DIAG[iN + NN*idiag] ) ; 
    }
    fprintf( fp, "\n " );
  }
  fclose(fp);
}
