#include <stdlib.h>
#include <stdio.h>
#include "chain.h"
#include "model.h"
/*=======================================*/
void alloc_chain( double **PSI, double **psi){
  double *temp;
  *PSI = (double *) malloc( ndim*sizeof( double ) );
  hipMalloc( (void**)&temp, ndim*sizeof( double ) );
  *psi = temp;
}
/*-------------------------------------------------------------------*/
void  free_chain( double **PSI, double **psi  ){
  free( *PSI );
  hipFree( *psi );
}
/*-------------------------------------------------------------------*/
void H2D(double psi[], double PSI[], int Nsize){
  hipMemcpy( psi, PSI, Nsize*sizeof(double), hipMemcpyHostToDevice);
}
/*-------------------------------------------------------------------*/
void D2H(double PSI[], double psi[], int Nsize){
  hipMemcpy( PSI, psi, Nsize*sizeof(double), hipMemcpyDeviceToHost);
}
/*-------------------------------------------------------------------*/
void iniconf(  double PSI[], double psi[]){
  set_param( );
  initial_configuration( PSI ) ;
  H2D( psi, PSI, ndim );
}

