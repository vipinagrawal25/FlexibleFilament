#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <strings.h>
#include "evolve.h"
#include <math.h>
#define tiny pow(10,-15)
double *dev_kk;
double *dev_psip, *dev_k1, *dev_k2, *dev_k3, *dev_k4, *dev_k5, *dev_k6, **dev_kin;
double *dev_err;
double *dev_redux, *REDUX; 
int size_redux;
double MaxLen, MinLen;
double *dev_EForce;
using namespace std;
/*--------------------------------------------------*/
void (*ALGO)( double [], double [],
              double [], double [],
              EV* , EV *,
              MPARAM ,   MPARAM *,
              double [], double [],  
              CRASH , CRASH *,
              int , int  );
void euler( double PSI[], double dev_psi[],
            double VEL[], double dev_vel[],
            EV* TT, EV *dev_tt,
            MPARAM PARAM,   MPARAM *dev_param,
            double DIAG[], double dev_diag[],  
            CRASH BUG, CRASH *dev_bug,
            int Nblock, int Nthread );
void rnkt4( double PSI[], double dev_psi[],
            double VEL[], double dev_vel[],
            EV* TT, EV *dev_tt,
            MPARAM PARAM,   MPARAM *dev_param,
            double DIAG[], double dev_diag[],
            CRASH BUG, CRASH *dev_bug,
            int Nblock, int Nthread );
void rnkf45( double PSI[], double dev_psi[],
             double VEL[], double dev_vel[],
            EV* TT, EV *dev_tt,
            MPARAM PARAM,   MPARAM *dev_param,
            double DIAG[], double dev_diag[],
            CRASH BUG, CRASH *dev_bug,
            int Nblock, int Nthread );
__global__ void eval_rhs(double kk[], double psi[], double EForce[],
                         EV *tt, MPARAM *param, double diag[],
                         CRASH *bug );
void pre_euler( int Nsize);
void pre_rnkt4( int Nsize);
void pre_rnkf45( int Nsize, int Nblock, int Nthread );
void eu_time_step( EV* TT, EV *dev_tt );
// __global__ void reduce_diag( double diag[] );
__global__ void eu_psi_step( double psi[], double kk[], EV *tt );
__global__ void rnkf45_calc_error(  double error[], double *kptr[], EV *tt );
double SumDevArray(double dev_array[], int Nblock, int Nthread);
/*-----------------------------------------------------------------------------*/
void  post_euler( void  ){
  hipFree( dev_kk );
  hipFree( dev_EForce);
}
/*-----------------------------------------------------------------------------*/
void  post_rnkt4( void  ){
  hipFree( dev_psip );
  hipFree( dev_k1 );
  hipFree( dev_k2 ) ;
  hipFree( dev_k3 ) ;
  hipFree( dev_k4 );
  hipFree(dev_EForce);
}
/*------------------------------------------------------------------------------*/
void  post_rnkf45( void  ){
  hipFree( dev_psip );
  hipFree( dev_k1 );
  hipFree( dev_k2 ) ;
  hipFree( dev_k3 ) ;
  hipFree( dev_k4 );
  hipFree( dev_k5 );
  hipFree( dev_k6 );
  hipFree( dev_kin);
  hipFree( dev_redux );
  hipFree( dev_err );
  hipFree(dev_EForce);
}
/*-------------------------------------------------------------------------------*/
void post_evolve( char *algo  ){
  if ( strcmp( algo , "euler") == 0 ){
    post_euler( );
  } else if ( strcmp( algo, "rnkt4" ) == 0 ){
    post_rnkt4( );
  } else if ( strcmp( algo, "rnkf45" ) == 0){
    post_rnkf45( );
  }
  else {
    printf( " algorithm\t%s\t not coded \n", algo);
    printf( "EXITING \n " );
    exit(1);
  }
}
/*----------------------------------------------------------------------------------*/
void pre_evolve( int Nsize, char *algo, EV *TT,  EV **dev_tt, int Nblock, int Nthread ){
  if ( strcmp( algo , "euler") == 0 ){
    pre_euler( Nsize );
    ALGO = &euler;
  } else if ( strcmp( algo, "rnkt4" ) == 0 ){
    pre_rnkt4( Nsize );
    ALGO = &rnkt4;
  } else if (strcmp( algo, "rnkf45" ) == 0){
    pre_rnkf45( Nsize, Nblock, Nthread );
    ALGO = &rnkf45;
  }
 else {
    printf( " algorithm\t%s\t not coded \n", algo);
    printf( "EXITING \n " );
    exit(1);
  }
  /* Set up parameters for evolution */
  /* Should we save them somewhere else? */
  /*Yes we should save them somewhere else.--vipin*/
  (*TT).time = 0.;            // current time of the simulation.
  (*TT).tprime = (*TT).time;  // time after every substep inside any function.
  (*TT).dt = 1.e-5;           // initial dt for the evolution
  (*TT).ndiag = 500;          // total number of files
  (*TT).tmax = 10;            // Tmax for the simulation.
  (*TT).tdiag = 0.;           // Time to save diagnostics
  (*TT).substep = 0.;         // substep inside the function calculation
  EV *temp ;
  hipMalloc(  (void**)&temp, size_EV );
  *dev_tt = temp;
  hipMemcpy( *dev_tt, TT,
                     size_EV, hipMemcpyHostToDevice);
  wevolve( *TT, "initial.txt");
}
/* ------------------------------------------------------------------------------*/
void wevolve( EV TT, char *fname ){
  FILE *pout ;
  pout = fopen ( fname, "w" );
  fprintf( pout, "# =========== Evolve Parameters==========\n" );
  fprintf( pout, " time=%lf \n ", TT.time ) ;
  fprintf( pout, " tprime=%lf \n ", TT.tprime ) ;
  fprintf( pout, "dt = %lf \n ", TT.dt );
  fprintf( pout, " ndiag = %d \n ", TT.ndiag );
  fprintf( pout, "tmax = %lf \n ", TT.tmax );
  fprintf( pout, "tdiag = %lf \n ", TT.tdiag );
}
  /*----------------------------------------*/
void pre_euler( int Nsize ){
  printf( " #---time-integration algorithm : EULER --\n " );
  hipMalloc( (void**)&dev_kk, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_EForce, Nsize*sizeof( double ) );
  printf( "#--I have set up auxiliary storage in the device-- \n " ) ;
}
/*----------------------------------------*/
void pre_rnkt4( int Nsize ){
  printf( "#-- time-integration algorithm : RNKT4-- \n " );
  hipMalloc( (void**)&dev_psip, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k1, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k2, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k3, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k4, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_EForce, Nsize*sizeof(double) );
  printf( "--I have set up auxiliary storage in the device --\n " ) ;
}
/*--------------------------------------------------------------------*/
void pre_rnkf45( int Nsize, int Nblock, int Nthread ){
  printf( "#-- time-integration algorithm : RNKF45-- \n " );
  hipMalloc( (void**)&dev_psip, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k1, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k2, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k3, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k4, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k5, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_k6, Nsize*sizeof( double ) );
  hipMalloc( (void**)&dev_err, NN*sizeof(double));
  hipMalloc( (void**)&dev_kin, 6*sizeof(double*));
  hipMalloc( (void**)&dev_EForce, Nsize*sizeof(double));
  // Defining array of all k pointers.
  double *KIN[6] = {dev_k1,dev_k2,dev_k3,dev_k4,dev_k5,dev_k6};
  hipMemcpy(dev_kin,KIN,6*sizeof(double*),hipMemcpyHostToDevice); 
  size_redux = Nblock*sizeof(double);
  hipMalloc( (void**)&dev_redux, size_redux );
  REDUX = (double *) malloc(size_redux);
  for (int iblock = 0; iblock < Nblock; ++iblock){
    REDUX[iblock] = 0.;
  }
  hipMemcpy(dev_redux,REDUX,size_redux,hipMemcpyHostToDevice);
  printf( "--I have set up auxiliary storage in the device --\n " ) ;
}
/*---------------------------------------------------------------------*/
void evolve( double PSI[], double dev_psi[], 
             double VEL[], double dev_vel[],
             EV *TT, EV *dev_tt,
             MPARAM PARAM, MPARAM *dev_param ,
             double DIAG[], double dev_diag[], int size_diag,
             CRASH BUG,  CRASH *dev_bug,
             int Nblock, int Nthread ) {
  MaxLen=1.;
  MinLen=1.;
  hipDeviceProp_t *prop;
  int count;
  qdevice( &count, &prop ) ;
  printf( "#- We know device properties \n");
// copy the time data to device. 
  hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice);
  while ( (*TT).time < (*TT).tmax){
      (*TT).ldiag = 0 ;
      // printf( "time=%lf\t #- dt=%lf\t tmax=%lf \n", (*TT).time, ((*TT).dt), (*TT).tmax ) ;
    if( (*TT).time >= (*TT).tdiag ) {
      (*TT).ldiag = 1;
      (*TT).tdiag = (*TT).time +  (*TT).tmax/((double) (*TT).ndiag) ;
      cout << "time=" << (*TT).time << "\tdt=" << (*TT).dt << "\t tmax="<< (*TT).tmax << endl;
    }
    hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice ) ;
    ALGO( PSI, dev_psi,
          VEL, dev_vel,
          TT, dev_tt,
          PARAM, dev_param ,
          DIAG, dev_diag,
          BUG, dev_bug,
          Nblock, Nthread );  
  } 
  cout << "Maximum length: " << MaxLen << "\t Minimum length: " << MinLen << endl; 
  // while time loop ends here
}
/*-------------------------------------------------------------------------*/
__global__ void eval_rhs(double kk[], double psi[], double EForce[],
                         EV *tt, MPARAM *param, double diag[],
                         CRASH *bug ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < NN){
    model_rhs(kk, psi, EForce, tid, (*tt).tprime, param, diag, bug, (*tt).ldiag);
    tid += blockDim.x * gridDim.x ;
  }// while loop over threads finishes here.
}
/*--------------------------------------------------------------------------- */
void euler( double PSI[], double dev_psi[],
            double VEL[], double dev_vel[],
            EV *TT, EV *dev_tt,
            MPARAM PARAM,   MPARAM *dev_param,
            double DIAG[], double dev_diag[],  
            CRASH BUG, CRASH *dev_bug,
            int Nblock, int Nthread ){
// evaluate the right hand side in a kernel 
  eval_rhs<<<Nblock,Nthread >>>( dev_kk, dev_psi, dev_EForce, dev_tt ,
                                 dev_param, dev_diag, dev_bug );
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  dev_vel = dev_kk;
  // reduce diagnostic
  if ( (*TT).ldiag ) {
    //reduce_diag<<<Nblock, Nthread >>> ( dev_diag ) ;  
    // diagnostic copied to host out here
    int size_diag = NN * PARAM.qdiag * sizeof(double) ;
    hipMemcpy( DIAG, dev_diag, size_diag, hipMemcpyDeviceToHost);
    wDIAG( DIAG, (*TT).time, PARAM );
    D2H( PSI, dev_psi, ndim);
    D2H(VEL, dev_vel, ndim);
    wPSI( PSI, VEL, (*TT).time ) ; 
  }
  // take the Euler step
  eu_psi_step<<< Nblock, Nthread >>>( dev_psi, dev_kk, dev_tt ) ;
  eu_time_step(TT, dev_tt);
 }
/*----------------------------------------------------------------*/
__global__ void eu_psi_step( double psi[], double kk[], EV *tt ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < NN ){
    for ( int ip=0; ip<pp; ip++){
      psi[ip+pp*tid] += kk[ip+pp*tid]*(*tt).dt ;
    }
    tid += blockDim.x * gridDim.x ;
  } // loop over threads ends here
}
/*-----------------------------------------------------------------------*/
void eu_time_step( EV *TT, EV *dev_tt ){
  (*TT).time = (*TT).time + (*TT).dt ;
  (*TT).tprime = (*TT).time ;
  (*TT).substep = 0;
  hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice ) ;
}
/*-----------------------------------------------------------------------*/
void rk4_time_substep( EV *TT, EV *dev_tt, int j ){
  double rk4a[4] ;
  rk4a[0] = 1./2. ;
  rk4a[1] = 1./2. ;
  rk4a[2] = 1. ;
  rk4a[3] = 0. ;
  (*TT).tprime = (*TT).time + ((*TT).dt)*rk4a[ j ] ;
  (*TT).substep = j + 1;
  (*TT).ldiag=0;
  hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice ) ;
}
/*----------------------------------------------------------------*/
__global__ void rk4_psi_substep( double psip[], double kin[], double psi[], EV *tt ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double rk4a[4] ;
  rk4a[0] = 1./2. ;
  rk4a[1] = 1./2. ;
  rk4a[2] = 1. ;
  rk4a[3] = 0. ;
  int j = (*tt).substep ;
  while (tid < NN ){
    for ( int ip=0; ip<pp; ip++){
      psip[ip+pp*tid] = psi[ip+pp*tid] + kin[ip+pp*tid]*(*tt).dt*rk4a[ j ] ;
    }
    tid += blockDim.x * gridDim.x ;
  } // loop over threads ends here
}
/*-----------------------------------------------------------------------*/
void rk4_time_step( EV *TT, EV *dev_tt ){
  (*TT).time = (*TT).time + (*TT).dt ;
  (*TT).tprime = (*TT).time ;
  (*TT).substep = 0;
  hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice );
}
/*----------------------------------------------------------------*/
__global__ void rk4_psi_step( double psi[],
                          double k1[], double k2[],
                          double k3[], double k4[],
                          EV *tt ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < NN ){
    for ( int ip=0; ip<pp; ip++){
      psi[ip+pp*tid] = psi[ip+pp*tid] +
        (k1[ip+pp*tid]/6.+ k2[ip+pp*tid]/3. + k3[ip+pp*tid]/3. + k4[ip+pp*tid]/6. )*(*tt).dt ;
    }
    tid += blockDim.x * gridDim.x ;
  } // loop over threads ends here
}
/*-----------------------------------------------------------------------*/
void rnkt4( double PSI[], double dev_psi[],
            double VEL[], double dev_vel[],
            EV* TT, EV *dev_tt,
            MPARAM PARAM, MPARAM *dev_param,
            double DIAG[], double dev_diag[],
            CRASH BUG, CRASH *dev_bug,
            int Nblock, int Nthread ){
  /* I do time-marching */
  // 1st evaluation of rhs, diagnostic is calculated in this step
  eval_rhs<<<Nblock,Nthread >>>( dev_k1, dev_psi, dev_EForce, dev_tt ,
                                 dev_param, dev_diag, dev_bug );
    // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  dev_vel = dev_k1;
 // reduce diagnostic
  if ( (*TT).ldiag ) {
    //reduce_diag<<<Nblock, Nthread >>> ( dev_diag ) ;  
    // diagnostic copied to host out here
    // printf( "calculating diagnostics \n " );
    int size_diag = NN * PARAM.qdiag * sizeof(double) ;
    hipMemcpy( DIAG, dev_diag, size_diag, hipMemcpyDeviceToHost);
    wDIAG( DIAG, (*TT).time, PARAM );
    D2H( PSI, dev_psi, ndim);
    D2H(VEL,dev_vel,ndim);
    wPSI( PSI, VEL, (*TT).time ) ; 
  }
  // take the first substep
  rk4_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_k1, dev_psi, dev_tt ) ;
  rk4_time_substep( TT, dev_tt , 0 ) ;
  // 2nd evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k2, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug);
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  // take the second substep
  rk4_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_k2, dev_psi, dev_tt ) ;
  rk4_time_substep( TT, dev_tt , 1 ) ;
  // 3rd evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k3, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug);
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  // take the third substep
  rk4_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_k3, dev_psi, dev_tt ) ;
  rk4_time_substep( TT, dev_tt , 2 ) ;
   // 4th evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k4, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug );
   // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  // final step
    rk4_psi_step<<<Nblock, Nthread >>>( dev_psi,
                                    dev_k1, dev_k2, dev_k3, dev_k4, dev_tt );
    rk4_time_step( TT, dev_tt ) ;
   }
/*----------------------------------------------------------------*/
__global__ void rnkf45_psi_substep( double psip[], double* kin[], double psi[], EV *tt ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //Cash-Karp parameter
  // double rnkf45b[5][5] = {{0.2,0.,0.,0.,0.},
  //                       {3./40,9./40,0.,0.,0.},
  //                       {0.3,-0.9,1.2,0.,0.},
  //                       {-11./54.,2.5,-70./27.,35./27.,0},
  //                       {1631./55296.,175./512.,575./13824.,44275./110592.,253./4096.}};

  //original fehlberg parameters
  double rnkf45b[5][5]= {{0.25,0,0,0,0},
                        {3./32.,9./32.,0,0,0},
                        {1932./2197.,-7200./2197.,7296./2197.,0.,0.},
                        {439./216.,-8.,3680./513.,-845./4104.,0},
                        {-8./27.,2.,-3544./2565.,1859./4104.,-11./40.}};

  int j = (*tt).substep ;
  // double *kk = {&kin[0],&kin[ndim],&kin[2*ndim],&kin[3*ndim],&kin[4*ndim],&kin[5*ndim]}
  while (tid < NN ){
    for ( int ip=0; ip<pp; ip++){
        psip[ip+pp*tid] = psi[ip+pp*tid];
        for (int jp = 0; jp < 6; ++jp)
        {
          psip[ip+pp*tid] += kin[jp][ip+pp*tid]*rnkf45b[j][jp]*(*tt).dt;
        }
    }
    tid += blockDim.x * gridDim.x ;
  } // loop over threads ends here
}
/*-----------------------------------------------------------------------*/
void rnkf45_time_substep( EV* TT, EV *dev_tt, int j ){
  // cash-karp parameter
  // double rnkf45a[5] = {0.2,0.3,0.6,1.,7./8} ;

  //original fehlberg parameters
  double rnkf45a[5] = {0.25,3./8,12./13,1.,1./2}; 
  (*TT).tprime = (*TT).time + ((*TT).dt)*rnkf45a[ j ] ;
  (*TT).substep = j + 1;
  (*TT).ldiag=0;
  hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice ) ;
}
/*----------------------------------------------------------------*/
__global__ void rnkf45_calc_error(  double error[], double *kptr[], EV *tt ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // cash-karp parameter
  // double rnkf45c[2][6] = { {2825./27648,0.,18575./48384,13525./55296,277./14336,0.25},
  //                            {37./378,0.,250./621,125./594,0.,512./1771} };

  // original fehlberg parameters
  double rnkf45c[2][6] = { {25./216.,0,1408./2565.,2197./4104.,-1./5.,0},
                          {16./135.,0,6656./12825.,28561./56430.,-9./50.,2./55.} };
  //
  error[tid]=0;
  double temp_error;
  while (tid < NN ){
    for ( int ip=0; ip<pp; ip++){
      temp_error=0.;
      for (int jp = 0; jp < 6; ++jp){
          temp_error += ((rnkf45c[0][jp]-rnkf45c[1][jp])*kptr[jp][ip+pp*tid])*((*tt).dt);
        // temp_error += tid;
      }
      temp_error = abs(temp_error);
      error[tid] = max(error[tid],temp_error);
      // error[tid]=temp_error; 
    }
    tid += blockDim.x * gridDim.x ;
  } // loop over threads ends here
  __syncthreads();
}
/*----------------------------------------------------------------*/
__global__ void rnkf45_psi_step( double psi[], double *kptr[], EV *tt ){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Cash-Karp parameters
  // double rnkf45c[6] = {37./378,0,250./621,125./594,0.,512./1771}; 

  // original fehlberg parameters
  double rnkf45c[6] = {16./135.,0.,6656./12825.,28561./56430.,-9./50.,2./55.};

  while (tid < NN ){
    for ( int ip=0; ip<pp; ip++){
      for (int jp = 0; jp < 6; ++jp){
          psi[ip+pp*tid] += (rnkf45c[jp]*kptr[jp][ip+pp*tid])*((*tt).dt);  
      }
    }
    tid += blockDim.x * gridDim.x ;
  } // loop over threads ends here
}
/*-----------------------------------------------------------------------*/
bool rnkf45_time_step( EV* TT, EV *dev_tt, double maxErr){
  // hipMemcpy(  &TT, dev_tt, size_EV, hipMemcpyDeviceToHost ) ;
  double tol = 1.e-6;
  double truncationmax=2;   // Maximum multiplication in time step
  double truncationmin=0.5; // Minimum multiplication in time step
  bool laccept;
  double s;
  double eps=0.84;        // Safety factor to avoid the infinite loop.
  maxErr=maxErr+tiny;     // in case maxErr is too small to be almost zero.
  if (maxErr<tol){
    laccept=1;
    (*TT).time = (*TT).time + (*TT).dt;
    (*TT).tprime = (*TT).time ;
    (*TT).substep = 0;
    s = eps*pow((tol/maxErr),0.25);
    // cout << s << endl;
    if (s>truncationmax){ s=truncationmax;}
    (*TT).dt=s*((*TT).dt);
  }
  else{
    laccept=0;  
    (*TT).tprime = (*TT).time ;
    (*TT).substep = 0;
    s = eps*pow((tol/maxErr),0.2);
    if (s<truncationmin){s=truncationmin;}
    (*TT).dt = s*((*TT).dt);
    (*TT).ldiag = 0;
    // printf( "## Rejected- tmax=%f\t time=%f\t dt=%f \n", (*TT).tmax, (*TT).time, (*TT).dt ) ;
  }
  hipMemcpy( dev_tt, TT, size_EV, hipMemcpyHostToDevice );
  return laccept;
}
/*----------------------------------------------------------------*/
double MaxDevArray(double dev_array[], int Nblock, int Nthread){
  double maxA=0.;
  for (int iblock = 0; iblock < Nblock; ++iblock){
    REDUX[iblock]=0.;
  }
  thread_maxima <<< Nblock, Nthread, Nthread*sizeof(double) >>> (dev_array,dev_redux);
  hipMemcpy(REDUX,dev_redux,size_redux,hipMemcpyDeviceToHost);
  // Copied the thread maxima output back to host.
  // Compare the maximum across the blocks. This operation is done in CPU for the time being.
  // Calculate maxima using STL
  // This could be done better by launching a kernel 
  for (int iblock = 0; iblock < Nblock; ++iblock){
    // maxA = max(maxA,REDUX[iblock]);
    maxA = max(maxA,REDUX[iblock]); 
    // cout << REDUX[iblock] << "\t" ;
  } 
  return maxA;
}
/*-----------------------------------------------------------------------*/
void rnkf45( double PSI[], double dev_psi[],
             double VEL[], double dev_vel[],
            EV* TT, EV *dev_tt,
            MPARAM PARAM,   MPARAM *dev_param,
            double DIAG[], double dev_diag[], 
            CRASH BUG, CRASH *dev_bug,
            int Nblock, int Nthread ){

  int size_diag = NN * PARAM.qdiag * sizeof(double) ;
  double StringLen;
  /* I do time-marching */
  // 1st evaluation of rhs, diagnostic is calculated in this step
  eval_rhs<<< Nblock,Nthread >>>( dev_k1, dev_psi, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug  );
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  dev_vel = dev_k1;
  // reduce diagnostic
  if ( (*TT).ldiag ) {
    //reduce_diag<<<Nblock, Nthread >>> ( dev_diag ) ;  
    // diagnostic copied to host out here
    // printf( "calculating diagnostics \n " );
    hipMemcpy( DIAG, dev_diag, size_diag,hipMemcpyDeviceToHost );
    wDIAG( DIAG, (*TT).time, PARAM );
    D2H( PSI, dev_psi, ndim );
    D2H(VEL,dev_vel,ndim);
    wPSI( PSI, VEL, (*TT).time ) ;
  }
// take the first substep
  rnkf45_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_kin, dev_psi, dev_tt ) ;
  // D2H(PSI, dev_psi, ndim);
  // wPSI( PSI, (*TT).time ) ; 
  rnkf45_time_substep( TT, dev_tt , 0 ) ;
  // 2nd evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k2, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug );
  // check if there were any bugs from rhs evaluation
  hipMemcpy(&BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  // take the second substep
  rnkf45_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_kin, dev_psi, dev_tt ) ;
  // D2H(PSI, dev_psi, ndim);
  // wPSI( PSI, (*TT).time ) ; 
  rnkf45_time_substep( TT, dev_tt , 1 ) ;
  // 3rd evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k3, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug );
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  // take the third substep
  rnkf45_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_kin, dev_psi, dev_tt ) ;
  // D2H(PSI, dev_psi, ndim);
  // wPSI( PSI, (*TT).time ) ; 
  rnkf45_time_substep( TT, dev_tt , 2 ) ;
  // 4th evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k4, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug);
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}

  // take the fourth substep
  rnkf45_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_kin, dev_psi, dev_tt ) ;
  // D2H(PSI, dev_psi, ndim);
  // wPSI( PSI, (*TT).time ) ; 
  rnkf45_time_substep( TT, dev_tt , 3) ;
  // 5th evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k5, dev_psip, dev_EForce,  dev_tt ,
                                 dev_param, dev_diag, dev_bug );
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}

  // take the fifth substep
  rnkf45_psi_substep<<<Nblock,Nthread>>>( dev_psip,  dev_kin, dev_psi, dev_tt ) ;
  // D2H(PSI, dev_psi, ndim);
  // wPSI( PSI, (*TT).time ) ;  
  rnkf45_time_substep( TT, dev_tt , 4) ;
  // 5th evaluation of rhs, no diagnostic calculated
  eval_rhs<<<Nblock,Nthread >>>( dev_k6, dev_psip, dev_EForce, dev_tt ,
                                 dev_param, dev_diag, dev_bug );
  // check if there were any bugs from rhs evaluation
  hipMemcpy( &BUG, dev_bug, size_CRASH, hipMemcpyDeviceToHost);
  if ( BUG.lstop) { IStop( BUG );}
  // final step
  rnkf45_calc_error<<<Nblock, Nthread >>>( dev_err, dev_kin, dev_tt );
  // psip will store the 4th order rnkt4 solution. 
  // rnkf45_psi_step<<<Nblock, Nthread >>>( dev_psip, dev_kin, dev_tt, 1 );
  // rnkf45_error<<Nblock,Nthread>>> (dev_err,dev_psi,dev_psip);
  double maxErr = MaxDevArray( dev_err, Nblock, Nthread );
  // cout << maxErr << endl;
  // For debugging stuff
  // printf("%lf\n",maxErr);
  bool laccept = rnkf45_time_step(TT,dev_tt,maxErr);
  if (laccept){
    // If the step is accepted -> this function calculates the PSI at next time step.
    rnkf45_psi_step<<<Nblock, Nthread >>>( dev_psi, dev_kin, dev_tt );
  }
  else{
    rnkf45( PSI, dev_psi,
            VEL, dev_vel,
          TT, dev_tt,
          PARAM, dev_param ,
          DIAG, dev_diag,
          BUG, dev_bug,
          Nblock, Nthread );
  }
}