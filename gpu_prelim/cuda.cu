#include <stdio.h>
#include "hip/hip_runtime.h"
/*========================================*/
void  qdevice(int *count, hipDeviceProp_t **prop ) {
  int ndevice;
  hipGetDeviceCount( &ndevice ) ;
  *prop = (hipDeviceProp_t *) malloc( ndevice*sizeof( hipDeviceProp_t ) );
  for (int i=0; i<ndevice; i++){
    hipGetDeviceProperties( prop[i], i);
  }
  *count = ndevice;
}
/*----------------------------------------------*/
  void qfree( hipDeviceProp_t *prop ){
    free( prop );
  }
/*----------------------------------------------*/
__device__ void scpy( char to[], char from[]){
  int i=0;
  while ( (to[i] = from[i]) != '\0')
    i = i+1;
}
/*----------------------------------------------*/
__device__ void device_exception( struct CRASH *bug, char *mesg ){
        (*bug).lstop = 1;
        scpy( (*bug).message, mesg ) ;
}
/*-----------------------------------------------------------------------------*/
void IStop( CRASH BUG ){
  printf( "#-I STOP, something went wrong \n") ;
  printf( "#-%s \n", BUG.message );
  exit(1);
}
/*-----------------------------------------------------------------------------*/
void set_crash( CRASH *BUG, CRASH **dev_bug ){
  (*BUG).lstop = 0;
  strcpy( (*BUG).message, " No bug yet" );
  CRASH *temp ;
  hipMalloc( (void**)&temp, size_CRASH );
  *dev_bug = temp;
  hipMemcpy( *dev_bug, BUG,
                     size_CRASH, hipMemcpyHostToDevice ) ;
} 
/*-----------------------------------------------------------------------------*/
