#include <stdio.h>
#include "hip/hip_runtime.h"
/*========================================*/
void  qdevice(int *count, hipDeviceProp_t **prop ) {
  int ndevice;
  hipGetDeviceCount( &ndevice ) ;
  *prop = (hipDeviceProp_t *) malloc( ndevice*sizeof( hipDeviceProp_t ) );
  for (int i=0; i<ndevice; i++){
    hipGetDeviceProperties( prop[i], i);
  }
  *count = ndevice;
}
  /*----------------------------------------------*/
  void qfree( hipDeviceProp_t *prop ){
    free( prop );
  }
