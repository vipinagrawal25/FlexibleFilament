#include <stdio.h>
#include "hip/hip_runtime.h"
/*========================================*/
void  qdevice(int *count, hipDeviceProp_t **prop ) {
  int ndevice;
  hipGetDeviceCount( &ndevice ) ;
  *prop = (hipDeviceProp_t *) malloc( ndevice*sizeof( hipDeviceProp_t ) );
  for (int i=0; i<ndevice; i++){
    hipGetDeviceProperties( prop[i], i);
  }
  *count = ndevice;
}
/*----------------------------------------------*/
  void qfree( hipDeviceProp_t *prop ){
    free( prop );
  }
/*----------------------------------------------*/
__device__ void scpy( char to[], char from[]){
  int i=0;
  while ( (to[i] = from[i]) != '\0')
    i = i+1;
}
/*----------------------------------------------*/

