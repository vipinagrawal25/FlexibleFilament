#include <stdio.h>
#include "hip/hip_runtime.h"
/*========================================*/
void  qdevice(int *count, hipDeviceProp_t **prop ) {
  int ndevice;
  hipGetDeviceCount( &ndevice ) ;
  *prop = (hipDeviceProp_t *) malloc( ndevice*sizeof( hipDeviceProp_t ) );
  for (int i=0; i<ndevice; i++){
    hipGetDeviceProperties( prop[i], i);
  }
  *count = ndevice;
}
/*----------------------------------------------*/
  void qfree( hipDeviceProp_t *prop ){
    free( prop );
  }
/*----------------------------------------------*/
__device__ void scpy( char to[], char from[]){
  int i=0;
  while ( (to[i] = from[i]) != '\0')
    i = i+1;
}
/*----------------------------------------------*/
__device__ void device_exception( struct CRASH *bug, char *mesg ){
        (*bug).lstop = 1;
        scpy( (*bug).message, mesg ) ;
}
/*-----------------------------------------------------------------------------*/
