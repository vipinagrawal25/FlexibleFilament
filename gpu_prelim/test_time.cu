#include <stdio.h>
#include <stdlib.h>
#include "evolve.h"
#include "model.h"
#include "hip/hip_runtime.h"
int main( void ){
    EV TT, *dev_tt;
    /*------------------------------------------------*/
  //alloc_chain( &PSI, &dev_psi );
    pre_evolve(  ndim, "euler" , &TT, &dev_tt  ) ;
     EV  TBACK;
  hipMemcpy( &TBACK, dev_tt,
                     size_EV, hipMemcpyDeviceToHost ) ;
  wevolve( &TBACK, "Tback.txt" );
}
