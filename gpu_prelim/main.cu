#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "evolve.h"
#include "model.h"
/*==============================================*/
int main( void ){
  CRASH BUG, *dev_bug ; 
  double *dev_psi, *PSI;
  MPARAM PARAM, *dev_param;
  double *dev_diag, *DIAG;
  EV TT, *dev_tt;
  int Nblock, Nthread;
  /*------------------------------------------------*/
  if ( NN < 128 ) {
    Nthread= 1;
    Nblock = NN ;
    } else{
    // Otherwise: we launch the threads differently:
    Nthread = 128;
    Nblock = (NN+127)/128;
  }
  printf( "#-I shall launch %d blocks, each with %d threads\n", Nblock, Nthread );
  /*------------------------------------------------*/

  set_crash(  &BUG, &dev_bug ); 
  alloc_chain( &PSI, &dev_psi );
  set_param( &PARAM, &dev_param ) ;
  int size_diag = pre_diag( &DIAG , &dev_diag, PARAM );
   // pre_evolve(  ndim, "euler" , &TT, &dev_tt, Nblock, Nthread  ) ;
  // pre_evolve(  ndim, "rnkt4" , &TT, &dev_tt, Nblock, Nthread  ) ;
  pre_evolve(  ndim, "rnkf45" , &TT, &dev_tt, Nblock, Nthread  ) ;
  // setup initial configuration 
  initial_configuration( PSI, PARAM ) ;
  wPSI( PSI, TT.time ) ; 
  H2D( dev_psi, PSI, ndim );
  printf( " #starting time evolution ...\n ");
  evolve( PSI, dev_psi, 
          &TT,  dev_tt,
          PARAM, dev_param ,
          DIAG, dev_diag, size_diag,
          BUG,  dev_bug, 
          Nblock, Nthread ) ;
  printf( "#... time evolution finished \n");
  H2D( dev_psi, PSI, ndim );
  wPSI( PSI, TT.time ) ; 
  free_chain( &PSI, &dev_psi );
}
