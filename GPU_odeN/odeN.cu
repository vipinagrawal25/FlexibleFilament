#include <stdlib.h>
#include <stdio.h>
#include "odeN.h"
#include "model.h"
/*=======================================*/
void alloc_odeN( double **PSI, double **psi){
  double *temp;
  *PSI = (double *) malloc( ndim*sizeof( double ) );
  hipMalloc( (void**)&temp, ndim*sizeof( double ) );
  *psi = temp;
}
/*-------------------------------------------------------------------*/
void  free_odeN( double **PSI, double **psi  ){
  free( *PSI );
  hipFree( *psi );
}
/*-------------------------------------------------------------------*/
void H2D(double psi[], double PSI[], int Nsize){
  hipMemcpy( psi, PSI, Nsize*sizeof(double), hipMemcpyHostToDevice);
}
/*-------------------------------------------------------------------*/
void D2H(double PSI[], double psi[], int Nsize){
  hipMemcpy( PSI, psi, Nsize*sizeof(double), hipMemcpyDeviceToHost);
}
/*-------------------------------------------------------------------*/
void iniconf(  double PSI[], double psi[]){
  set_param( );
  initial_configuration( PSI ) ;
  H2D( psi, PSI, ndim );
}

